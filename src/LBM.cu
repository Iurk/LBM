#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <dirent.h>
#include <errno.h>
#include <sys/stat.h>

#include "paths.h"
#include "LBM.h"
#include "dados.h"

using namespace myGlobals;

// Input data
__constant__ unsigned int q, Nx_d, Ny_d;
__constant__ double rho0_d, u_max_d, nu_d, tau_d;

//Lattice Data
__constant__ double cs_d, w0_d, wp_d, ws_d;
__device__ int *ex_d;
__device__ int *ey_d;

// Mesh data
__device__ bool *cylinder_d;
__device__ bool *fluid_d;

__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d){
	return (Nx_d*(Ny_d*(d) + y) + x);
}

__global__ void gpu_init_equilibrium(double*, double*, double*, double*);
__global__ void gpu_stream_collide_save(double*, double*, double*, double*, double*, double*, bool);
__global__ void gpu_compute_flow_properties(unsigned int, double*, double*, double*, double*);
__global__ void gpu_print_mesh(int);
__global__ void gpu_initialization(double*, double);

// Boundary Conditions
__device__ void gpu_zou_he_inlet(unsigned int x, unsigned int y, double *f, double *f1,
								double *f5, double *f8, double *r, double *u, double *v){

	double ux = u_max_d;
	double uy = 0;

	unsigned int idx_0 = gpu_fieldn_index(x, y, 0);
	unsigned int idx_2 = gpu_fieldn_index(x, y, 2);
	unsigned int idx_3 = gpu_fieldn_index(x, y, 3);
	unsigned int idx_4 = gpu_fieldn_index(x, y, 4);
	unsigned int idx_6 = gpu_fieldn_index(x, y, 6);
	unsigned int idx_7 = gpu_fieldn_index(x, y, 7);

	double rho = (f[idx_0] + f[idx_2] + f[idx_4] + 2*(f[idx_3] + f[idx_6] + f[idx_7]))/(1.0 - ux);
	*f1 = f[idx_3] + 2.0/3.0*rho*ux;
	*f5 = f[idx_7] - 0.5*(f[idx_2] - f[idx_4]) + 1.0/6.0*rho*ux;
	*f8 = f[idx_6] + 0.5*(f[idx_2] - f[idx_4]) + 1.0/6.0*rho*ux;

	*r = rho;
	*u = ux;
	*v = uy;
}

__device__ void gpu_outflow(unsigned int x, unsigned int y, unsigned int x_before, unsigned int y_before, double *f){

	f[gpu_fieldn_index(x, y, 0)] = f[gpu_fieldn_index(x_before, y_before, 0)];
	f[gpu_fieldn_index(x, y, 1)] = f[gpu_fieldn_index(x_before, y_before, 1)];
	f[gpu_fieldn_index(x, y, 2)] = f[gpu_fieldn_index(x_before, y_before, 2)];
	f[gpu_fieldn_index(x, y, 3)] = f[gpu_fieldn_index(x_before, y_before, 3)];
	f[gpu_fieldn_index(x, y, 4)] = f[gpu_fieldn_index(x_before, y_before, 4)];
	f[gpu_fieldn_index(x, y, 5)] = f[gpu_fieldn_index(x_before, y_before, 5)];
	f[gpu_fieldn_index(x, y, 6)] = f[gpu_fieldn_index(x_before, y_before, 6)];
	f[gpu_fieldn_index(x, y, 7)] = f[gpu_fieldn_index(x_before, y_before, 7)];
	f[gpu_fieldn_index(x, y, 8)] = f[gpu_fieldn_index(x_before, y_before, 8)];

}

__device__ void gpu_bounce_back(unsigned int x, unsigned int y, double *f2){
	unsigned int noslip[] = {0, 3, 4, 1, 2, 7, 8, 5, 6};

	for(int n = 1; n < q; ++n){
		unsigned int x_next = x + ex_d[n];
		unsigned int y_next = y + ey_d[n];

		bool solid = cylinder_d[gpu_scalar_index(x_next, y_next)];

		unsigned int noslip_n = noslip[n];
		if (solid){
			f2[gpu_fieldn_index(x, y, noslip_n)] = f2[gpu_fieldn_index(x, y, n)];
		}
	}
}

__device__ void gpu_equilibrium(unsigned int x, unsigned int y, double rho, double ux, double uy, double *feq){

	double cs2 = cs_d*cs_d;
	double cs4 = cs2*cs2;
	double cs6 = cs4*cs2;

	double A = 1.0/(cs2);
	double B = 1.0/(2.0*cs4);

	double W[] = {w0, wp, wp, wp, wp, ws, ws, ws, ws};
	for(int n = 0; n < q; ++n){
		
		double ux2 = ux*ux;
		double uy2 = uy*uy;
		double ex2 = ex_d[n]*ex_d[n];
		double ey2 = ey_d[n]*ey_d[n];

		double order_1 = A*(ux*ex_d[n] + uy*ey_d[n]);
		double order_2 = B*(ux2*(ex2 - cs2) + 2*ux*uy*ex_d[n]*ey_d[n] + uy2*(ey2 - cs2));

		feq[gpu_fieldn_index(x, y, n)] = W[n]*rho*(1 + order_1 + order_2);
	}
}

__device__ void gpu_nonequilibrium(unsigned int x, unsigned int y, double tauxx, double tauxy, double tauyy, double *fneq){

	double cs2 = cs_d*cs_d;
	double cs4 = cs2*cs2;

	double B = 1.0/(2.0*cs4);

	double W[] = {w0, wp, wp, wp, wp, ws, ws, ws, ws};
	for(int n = 0; n < q; ++n){
		
		double ex2 = ex_d[n]*ex_d[n];
		double ey2 = ey_d[n]*ey_d[n];

		double order_1 = B*(tauxx*(ex2 - cs2) + 2*tauxy*ex_d[n]*ey_d[n] + tauyy*(ey2 - cs2));

		fneq[gpu_fieldn_index(x, y, n)] = W[n]*(order_1);
	}	

}

__host__ void init_equilibrium(double *f1, double *r, double *u, double *v){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_init_equilibrium<<< grid, block >>>(f1, r, u, v);
	getLastCudaError("gpu_init_equilibrium kernel error");
}

__global__ void gpu_init_equilibrium(double *f1, double *r, double *u, double *v){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	gpu_equilibrium(x, y, rho, ux, uy, f1);
}

__host__ void stream_collide_save(double *f1, double *f2, double *feq, double *fneq, double *r, double *u, double *v, bool save){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	//dim3 grid(1,1,1);
	//dim3 block(1,1,1);

	gpu_stream_collide_save<<< grid, block >>>(f1, f2, feq, fneq, r, u, v, save);
	getLastCudaError("gpu_stream_collide_save kernel error");
}

__global__ void gpu_stream_collide_save(double *f1, double *f2, double *feq, double *fneq, double *r, double *u, double *v, bool save){

	const double omega = 1.0/tau_d;

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int x_att, y_att;

	double rho = 0, ux_i = 0, uy_i = 0;
	for(int n = 0; n < q; ++n){
		rho += f1[gpu_fieldn_index(x, y, n)];
		ux_i += f1[gpu_fieldn_index(x, y, n)]*ex_d[n];
		uy_i += f1[gpu_fieldn_index(x, y, n)]*ey_d[n];
	}

	double ux = ux_i/rho;
	double uy = uy_i/rho;

	r[gpu_scalar_index(x, y)] = rho;
	u[gpu_scalar_index(x, y)] = ux;
	v[gpu_scalar_index(x, y)] = uy;
	
	double cs2 = cs_d*cs_d;
	double cs4 = cs2*cs2

	gpu_equilibrium(x, y, rho, ux, uy, feq);

	// Approximation of fneq
	for(int n = 0; n < q; ++n){
		fneq[gpu_fieldn_index(x, y, n)] = f1[gpu_fieldn_index(x, y, n)] - feq[gpu_fieldn_index(x, y, n)];
	}

	// Calculating the Viscous stress tensor
	double tauxx = 0, tauxy = 0, tauyy = 0;
	for(int n = 0; n < q; ++n){
		tauxx += fneq[gpu_fieldn_index(x, y, n)]*ex_d[n]*ex_d[n];
		tauxy += fneq[gpu_fieldn_index(x, y, n)]*ex_d[n]*ey_d[n];
		tauyy += fneq[gpu_fieldn_index(x, y, n)]*ey_d[n]*ey_d[n];
	}

	// Recalculating fneq
	gpu_nonequilibrium(x, y, tauxx, tauxy, tauyy, fneq);

	// Collision Step
	for(int n = 0; n < q; ++n){
		f1[gpu_fieldn_index(x, y, n)] = feq[gpu_fieldn_index(x, y, n)] (1.0 - omega)*fneq[gpu_fieldn_index(x, y, n)];
	}

	// Stream Step
	for(int n = 0; n < q; ++n){
		x_att = (x + ex_d[n] + Nx_d)%Nx_d;
		y_att = (y + ey_d[n] + Ny_d)%Ny_d;

		f2[gpu_fieldn_index(x_att, y_att, n)] = f1[gpu_fieldn_index(x, y, n)];
	}



	bool node_fluid = fluid_d[gpu_scalar_index(x, y)];

	if (node_fluid){
		gpu_bounce_back(x, y, f2);
	}

	unsigned int idx_s = gpu_scalar_index(x, y);

	if(x == 0){
		unsigned int idx_1 = gpu_fieldn_index(x, y, 1);
		unsigned int idx_5 = gpu_fieldn_index(x, y, 5);
		unsigned int idx_8 = gpu_fieldn_index(x, y, 8);

		gpu_zou_he_inlet(x, y, f2, &f2[idx_1], &f2[idx_5], &f2[idx_8], &r[idx_s], &u[idx_s], &v[idx_s]);
	}

	if(x == Nx_d-1){

		int x_before = x - 1;
		gpu_outflow(x, y, x_before, y, f2);
	}

	if(y == 0){

		//int y_before = y + 1;
		//gpu_outflow(x, y, x, y_before, f0, f2);

		f2[gpu_fieldn_index(x, y, 2)] = f2[gpu_fieldn_index(x, y, 4)];
		f2[gpu_fieldn_index(x, y, 5)] = f2[gpu_fieldn_index(x, y, 7)];
		f2[gpu_fieldn_index(x, y, 6)] = f2[gpu_fieldn_index(x, y, 8)];
	}

	if(y == Ny_d-1){

		//int y_before = y - 1;
		//gpu_outflow(x, y, x, y_before, f0, f2);

		f2[gpu_fieldn_index(x, y, 4)] = f2[gpu_fieldn_index(x, y, 2)];
		f2[gpu_fieldn_index(x, y, 7)] = f2[gpu_fieldn_index(x, y, 5)];
		f2[gpu_fieldn_index(x, y, 8)] = f2[gpu_fieldn_index(x, y, 6)];
	}
}

__host__ double report_convergence(unsigned int t, double *u, double *u_old, double *conv_host, double *conv_gpu, bool msg){

	double conv;
	conv = compute_convergence(u, u_old, conv_host, conv_gpu);

	if(msg){
		std::cout << std::setw(10) << t << std::setw(20) << conv << std::endl;
	}

	return conv;
}

__host__ double compute_convergence(double *u, double *u_old, double *conv_host, double *conv_gpu){

	dim3 grid(1, Ny/nThreads, 1);
	dim3 block(1, nThreads, 1);

	gpu_compute_convergence<<< grid, block >>>(u, u_old, conv_gpu);
	getLastCudaError("gpu_compute_convergence kernel error");

	size_t conv_size_bytes = 2*grid.x*grid.y*sizeof(double);
	checkCudaErrors(hipMemcpy(conv_host, conv_gpu, conv_size_bytes, hipMemcpyDeviceToHost));

	double convergence;
	double sumuxe2 = 0.0
	double sumuxa2 = 0.0

	for(unsigned int i = 0; i < grid.x*grid.y; ++i){
		sumuxe2 += conv_host[2*i];
		sumuxa2 += conv_host[2*i+1];
	}

	convergence = sqrt(sumuxe2/sumuxa2);
	return convergence;
}

__global__ void gpu_compute_convergence(double *u, double *u_old, double *conv){

	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int x = Nx_d/4;

	extern __shared__ double data[];

	double *uxe2 = data;
	double *uxa2 = data + 1*blockDim.x;

	double ux = u[gpu_scalar_index(x, y)];
	double ux_old = u_old[gpu_scalar_index(x, y)];

	uxe2[threadIdx.y] = (ux - ux_old)*(ux - ux_old);
	uxa2[threadIdx.y] = ux_old*ux_old;

	__syncthreads();

	if(threadIdx.y == 0){

		size_t idx = 2*(gridDim.x*blockIdx.y + blockIdx.x);

		for(int n = 0; n < 2; ++n){
			conv[idx+n] = 0.0;
		}

		for(int i = 0; i < blockDim.x; ++i){
			conv[idx  ] += uxe2[i];
			conv[idx+1] += uxa2[i];
		}
	}
}

__host__ std::vector<double> report_flow_properties(unsigned int t, double conv, double *rho, double *ux, double *uy, double *prop_gpu, double *prop_host, bool msg){

	std::vector<double> prop;

	if(msg){
		prop = compute_flow_properties(t, rho, ux, uy, prop, prop_gpu, prop_host);
		std::cout << std::setw(10) << t << std::setw(13) << prop[0] << std::setw(15) << prop[1] << std::setw(20) << conv << std::endl;
	}

	return prop;
}

__host__ std::vector<double> compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop, double *prop_gpu, double *prop_host){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_compute_flow_properties<<< grid, block, block.x*sizeof(double) >>>(t, r, u, v, prop_gpu);
	getLastCudaError("gpu_compute_flow_properties kernel error");

	size_t prop_size_bytes = grid.x*grid.y*sizeof(double);
	checkCudaErrors(hipMemcpy(prop_host, prop_gpu, prop_size_bytes, hipMemcpyDeviceToHost));

	double E = 0.0;
	double sumuxe2 = 0.0
	double sumuxa2 = 0.0

	for(unsigned int i = 0; i < grid.x*grid.y; ++i){
		E += prop_host[i];
	}

	prop.push_back(E);

	return prop;
}

__global__ void gpu_compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop_gpu){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	extern __shared__ double data[];

	double *E = data;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	E[threadIdx.x] = rho*(ux*ux + uy*uy);

	__syncthreads();

	if (threadIdx.x == 0){
		
		size_t idx = 1*(gridDim.x*blockIdx.y + blockIdx.x);

		for(int n = 0; n < 1; ++n){
			prop_gpu[idx+n] = 0.0;
		}

		for(int i = 0; i < blockDim.x; ++i){
			prop_gpu[idx] += E[i];
		}
	}
}

void wrapper_input(unsigned int *nx, unsigned int *ny, double *rho, double *u, double *nu, const double *tau){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Nx_d), nx, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Ny_d), ny, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(rho0_d), rho, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(u_max_d), u, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nu_d), nu, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), tau, sizeof(double)));
}

void wrapper_lattice(unsigned int *ndir, double *c, double *w_0, double *w_s, double *w_d){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(q), ndir, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cs_d), c, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(w0_d), w_0, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wp_d), w_s, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ws_d), w_d, sizeof(double)));
}

__host__ int* generate_e(int *e, std::string mode){

	int *temp_e;

	size_t mem_e = ndir*sizeof(int);

	checkCudaErrors(hipMalloc(&temp_e, mem_e));
	checkCudaErrors(hipMemcpy(temp_e, e, mem_e, hipMemcpyHostToDevice));

	if(mode == "x"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ex_d), &temp_e, sizeof(temp_e)));
	}
	else if(mode == "y"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ey_d), &temp_e, sizeof(temp_e)));
	}

	return temp_e;
}

__host__ bool* generate_mesh(bool *mesh, std::string mode){

	int mode_num;
	bool *temp_mesh;

	checkCudaErrors(hipMalloc(&temp_mesh, mem_mesh));
	checkCudaErrors(hipMemcpy(temp_mesh, mesh, mem_mesh, hipMemcpyHostToDevice));
	

	if(mode == "solid"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cylinder_d), &temp_mesh, sizeof(temp_mesh)));
		mode_num = 1;
	}
	else if(mode == "fluid"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fluid_d), &temp_mesh, sizeof(temp_mesh)));
		mode_num = 1;
	}

	if(meshprint){
		gpu_print_mesh<<< 1, 1 >>>(mode_num);
		printf("\n");
	}

	return temp_mesh;
}

__global__ void gpu_print_mesh(int mode){
	if(mode == 1){
		for(int y = 0; y < Ny_d; ++y){
			for(int x = 0; x < Nx_d; ++x){
				printf("%d ", cylinder_d[Nx_d*y + x]);
			}
		printf("\n");
		}
	}
	else if(mode == 2){
		for(int y = 0; y < Ny_d; ++y){
			for(int x = 0; x < Nx_d; ++x){
				printf("%d ", fluid_d[Nx_d*y + x]);
			}
		printf("\n");
		}
	}
}

__host__ void initialization(double *array, double value){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_initialization<<< grid, block >>>(array, value);
	getLastCudaError("gpu_print_array kernel error");
}

__global__ void gpu_initialization(double *array, double value){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	array[gpu_scalar_index(x, y)] = value;
}

__host__ bool* create_pinned_mesh(bool *array){

	bool *pinned;
	const unsigned int bytes = Nx*Ny*sizeof(bool);

	checkCudaErrors(hipHostMalloc((void**)&pinned, bytes));
	memcpy(pinned, array, bytes);
	return pinned;
}

__host__ double* create_pinned_double(){

	double *pinned;

	checkCudaErrors(hipHostMalloc((void**)&pinned, mem_size_scalar));
	return pinned;
}
