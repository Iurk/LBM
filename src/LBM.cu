#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <dirent.h>
#include <errno.h>
#include <sys/stat.h>

#include "paths.h"
#include "LBM.h"
#include "dados.h"

using namespace myGlobals;

// Input data
__constant__ unsigned int q, Nx_d, Ny_d;
__constant__ double rho0_d, u_max_d, nu_d, tau_d;

//Lattice Data
__constant__ double cs_d, w0_d, ws_d, wd_d;
__device__ int *ex_d;
__device__ int *ey_d;

// Mesh data
__device__ bool *cylinder_d;
__device__ bool *fluid_d;

__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d){
	return (Nx_d*(Ny_d*(d - 1) + y) + x);
}

__global__ void gpu_init_equilibrium(double*, double*, double*, double*, double*);
__global__ void gpu_stream_collide_save(double*, double*, double*, double*, double*, double*, double*, double*, bool);
__global__ void gpu_compute_flow_properties(unsigned int, double*, double*, double*, double*);
__global__ void gpu_print_mesh(int);
__global__ void gpu_initialization(double*, double);

// Boundary Conditions
__device__ void gpu_zou_he_inlet(unsigned int x, unsigned int y, double *f0, double *f, double *f1,
								double *f5, double *f8, double *r, double *u, double *v){

	double ux = u_max_d;
	double uy = 0;

	unsigned int idx_0 = gpu_field0_index(x, y);
	unsigned int idx_2 = gpu_fieldn_index(x, y, 2);
	unsigned int idx_3 = gpu_fieldn_index(x, y, 3);
	unsigned int idx_4 = gpu_fieldn_index(x, y, 4);
	unsigned int idx_6 = gpu_fieldn_index(x, y, 6);
	unsigned int idx_7 = gpu_fieldn_index(x, y, 7);

	double rho = (f0[idx_0] + f[idx_2] + f[idx_4] + 2*(f[idx_3] + f[idx_6] + f[idx_7]))/(1.0 - ux);
	*f1 = f[idx_3] + 2.0/3.0*rho*ux;
	*f5 = f[idx_7] - 0.5*(f[idx_2] - f[idx_4]) + 1.0/6.0*rho*ux;
	*f8 = f[idx_6] + 0.5*(f[idx_2] - f[idx_4]) + 1.0/6.0*rho*ux;

	*r = rho;
	*u = ux;
	*v = uy;
}

__device__ void gpu_outflow(unsigned int x, unsigned int y, unsigned int x_before, unsigned int y_before, double *f0, double *f){

	f0[gpu_field0_index(x, y)] = f0[gpu_field0_index(x_before, y_before)];
	f[gpu_fieldn_index(x, y, 1)] = f[gpu_fieldn_index(x_before, y_before, 1)];
	f[gpu_fieldn_index(x, y, 2)] = f[gpu_fieldn_index(x_before, y_before, 2)];
	f[gpu_fieldn_index(x, y, 3)] = f[gpu_fieldn_index(x_before, y_before, 3)];
	f[gpu_fieldn_index(x, y, 4)] = f[gpu_fieldn_index(x_before, y_before, 4)];
	f[gpu_fieldn_index(x, y, 5)] = f[gpu_fieldn_index(x_before, y_before, 5)];
	f[gpu_fieldn_index(x, y, 6)] = f[gpu_fieldn_index(x_before, y_before, 6)];
	f[gpu_fieldn_index(x, y, 7)] = f[gpu_fieldn_index(x_before, y_before, 7)];
	f[gpu_fieldn_index(x, y, 8)] = f[gpu_fieldn_index(x_before, y_before, 8)];

}

__device__ void gpu_bounce_back(unsigned int x, unsigned int y, double *f2){
	unsigned int noslip[] = {0, 3, 4, 1, 2, 7, 8, 5, 6};

	for(int n = 1; n < q; ++n){
		unsigned int x_next = x + ex_d[n];
		unsigned int y_next = y + ey_d[n];

		bool solid = cylinder_d[gpu_scalar_index(x_next, y_next)];

		unsigned int noslip_n = noslip[n];
		if (solid){
			f2[gpu_fieldn_index(x, y, noslip_n)] = f2[gpu_fieldn_index(x, y, n)];
		}
	}
}

__host__ void init_equilibrium(double *f0, double *f1, double *r, double *u, double *v){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_init_equilibrium<<< grid, block >>>(f0, f1, r, u, v);
	getLastCudaError("gpu_init_equilibrium kernel error");
}

__global__ void gpu_init_equilibrium(double *f0, double *f1, double *r, double *u, double *v){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	double A = 1.0/(cs_d*cs_d);
	double B = 1.0/(2.0*cs_d*cs_d);

	double w0r = w0_d*rho;
	double wsr = ws_d*rho;
	double wdr = wd_d*rho;
	double omusq = 1.0 - B*(ux*ux + uy*uy);

	double Wrho[] = {w0r, wsr, wsr, wsr, wsr, wdr, wdr, wdr, wdr};

	f0[gpu_field0_index(x, y)] = Wrho[0]*(omusq);
	for(int n = 1; n < q; ++n){
		double eidotu = ux*ex_d[n] + uy*ey_d[n];
		f1[gpu_fieldn_index(x, y, n)] = Wrho[n]*(omusq + A*eidotu*(1.0 + B*eidotu));
	}
}

__host__ void stream_collide_save(double *f0, double *f1, double *f2, double *f0neq, double *f1neq, double *r, double *u, double *v, bool save){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	//dim3 grid(1,1,1);
	//dim3 block(1,1,1);

	gpu_stream_collide_save<<< grid, block >>>(f0, f1, f2, f0neq, f1neq, r, u, v, save);
	getLastCudaError("gpu_stream_collide_save kernel error");
}

__global__ void gpu_stream_collide_save(double *f0, double *f1, double *f2, double *f0neq, double *f1neq, double *r, double *u, double *v, bool save){

	const double omega = 1.0/tau_d;

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int xf = (x + 1)%Nx_d;		// Forward
	unsigned int yf = (y + 1)%Ny_d;		// Forward
	unsigned int xb = (Nx_d + x - 1)%Nx_d;	// Backward
	unsigned int yb = (Ny_d + y - 1)%Ny_d; // Backward

	double ft0 = f0[gpu_field0_index(x, y)];

	// Streaming step
	double ft1 = f1[gpu_fieldn_index(xb, y, 1)];
	double ft2 = f1[gpu_fieldn_index(x, yb, 2)];
	double ft3 = f1[gpu_fieldn_index(xf, y, 3)];
	double ft4 = f1[gpu_fieldn_index(x, yf, 4)];
	double ft5 = f1[gpu_fieldn_index(xb, yb, 5)];
	double ft6 = f1[gpu_fieldn_index(xf, yb, 6)];
	double ft7 = f1[gpu_fieldn_index(xf, yf, 7)];
	double ft8 = f1[gpu_fieldn_index(xb, yf, 8)];

	double f[] = {ft0, ft1, ft2, ft3, ft4, ft5, ft6, ft7, ft8};

	double rho = 0, ux_i = 0, uy_i = 0;

	for(int n = 0; n < q; ++n){
		rho += f[n];
		ux_i += f[n]*ex_d[n];
		uy_i += f[n]*ey_d[n];
	}

	double rhoinv = 1.0/rho;

	double ux = rhoinv*ux_i;
	double uy = rhoinv*uy_i;

	if(save){
		r[gpu_scalar_index(x, y)] = rho;
		u[gpu_scalar_index(x, y)] = ux;
		v[gpu_scalar_index(x, y)] = uy;
	}

	double A = 1.0/(cs_d*cs_d);
	double B = 1.0/(2.0*cs_d*cs_d);

	double w0r = w0_d*rho;
	double wsr = ws_d*rho;
	double wdr = wd_d*rho;

	double W[] = {w0_d, ws_d, ws_d, ws_d, ws_d, wd_d, wd_d, wd_d, wd_d};
	double Wrho[] = {w0r, wsr, wsr, wsr, wsr, wdr, wdr, wdr, wdr};

	double omusq = 1.0 - B*(ux*ux + uy*uy);

	// Approximation of fneq
	f0neq[gpu_field0_index(x, y)] = f[0] - Wrho[0]*omusq;
	for(int n = 1; n < q; ++n){
		double eidotu = ux*ex_d[n] + uy*ey_d[n];
		double feq = Wrho[n]*(omusq + A*eidotu*(1.0 + B*eidotu));
		f1neq[gpu_fieldn_index(x, y, n)] = f[n] - feq;
	}

	// Calculating the Viscous stress tensor
	double tauxx = 0, tauxy = 0, tauyy = 0;
	for(int n = 1; n < q; ++n){
		tauxx += f1neq[gpu_fieldn_index(x, y, n)]*ex_d[n]*ex_d[n];
		tauxy += f1neq[gpu_fieldn_index(x, y, n)]*ex_d[n]*ey_d[n];
		tauyy += f1neq[gpu_fieldn_index(x, y, n)]*ey_d[n]*ey_d[n];
	}

	f0[gpu_field0_index(x, y)] = (1.0 - omega)*f0neq[gpu_field0_index(x, y)] + Wrho[0]*(omusq);

	for(int n = 1; n < q; ++n){
		f1neq[gpu_fieldn_index(x, y, n)] = B*W[n]*(tauxx*(A*ex_d[n]*ex_d[n] - 1.0) + 2.0*tauxy*A*ex_d[n]*ey_d[n] + tauyy*(A*ey_d[n]*ey_d[n] - 1.0));
		double eidotu = ux*ex_d[n] + uy*ey_d[n];
		double feq = Wrho[n]*(omusq + A*eidotu*(1.0 + B*eidotu));
		f2[gpu_fieldn_index(x, y, n)] = (1.0 - omega)*f1neq[gpu_fieldn_index(x, y, n)] + feq;
	}

	bool node_fluid = fluid_d[gpu_scalar_index(x, y)];

	if (node_fluid){
		gpu_bounce_back(x, y, f2);
	}

	unsigned int idx_s = gpu_scalar_index(x, y);

	if(x == 0){
		unsigned int idx_1 = gpu_fieldn_index(x, y, 1);
		unsigned int idx_5 = gpu_fieldn_index(x, y, 5);
		unsigned int idx_8 = gpu_fieldn_index(x, y, 8);

		gpu_zou_he_inlet(x, y, f0, f2, &f2[idx_1], &f2[idx_5], &f2[idx_8], &r[idx_s], &u[idx_s], &v[idx_s]);
	}

	if(x == Nx_d-1){

		int x_before = x - 1;
		gpu_outflow(x, y, x_before, y, f0, f2);
	}

	if(y == 0){

		//int y_before = y + 1;
		//gpu_outflow(x, y, x, y_before, f0, f2);

		f2[gpu_fieldn_index(x, y, 2)] = f2[gpu_fieldn_index(x, y, 4)];
		f2[gpu_fieldn_index(x, y, 5)] = f2[gpu_fieldn_index(x, y, 7)];
		f2[gpu_fieldn_index(x, y, 6)] = f2[gpu_fieldn_index(x, y, 8)];
	}

	if(y == Ny_d-1){

		//int y_before = y - 1;
		//gpu_outflow(x, y, x, y_before, f0, f2);

		f2[gpu_fieldn_index(x, y, 4)] = f2[gpu_fieldn_index(x, y, 2)];
		f2[gpu_fieldn_index(x, y, 7)] = f2[gpu_fieldn_index(x, y, 5)];
		f2[gpu_fieldn_index(x, y, 8)] = f2[gpu_fieldn_index(x, y, 6)];
	}
}

__host__ void compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop, double *prop_gpu, double *prop_host){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_compute_flow_properties<<< grid, block, block.x*sizeof(double) >>>(t, r, u, v, prop_gpu);
	getLastCudaError("gpu_compute_flow_properties kernel error");

	size_t prop_size_bytes = grid.x*grid.y*sizeof(double);
	checkCudaErrors(hipMemcpy(prop_host, prop_gpu, prop_size_bytes, hipMemcpyDeviceToHost));

	double E = 0.0;

	for(unsigned int i = 0; i < grid.x*grid.y; ++i){

		E += prop_host[i];
	}

	prop[0] = E;
}

__global__ void gpu_compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop_gpu){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	extern __shared__ double data[];

	double *E = data;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	E[threadIdx.x] = rho*(ux*ux + uy*uy);

	__syncthreads();

	if (threadIdx.x == 0){
		
		size_t idx = 1*(gridDim.x*blockIdx.y + blockIdx.x);

		for(int n = 0; n < 1; ++n){
			prop_gpu[idx+n] = 0.0;
		}

		for(int i = 0; i < blockDim.x; ++i){
			prop_gpu[idx] += E[i];
		}
	}
}

__host__ void report_flow_properties(unsigned int t, double *rho, double *ux, double *uy,
									 double *prop_gpu, double *prop_host){

	double prop[1];
	compute_flow_properties(t, rho, ux, uy, prop, prop_gpu, prop_host);
	printf("%u, %g\n", t, prop[0]);
}

__host__ void save_scalar(const std::string name, double *scalar_gpu, double *scalar_host, unsigned int n){

	std::ostringstream path, filename;

	std::string ext = ".dat";

	int ndigits = floor(log10((double)NSTEPS) + 1.0);

	const char* path_results_c = strdup(folder.c_str());

	DIR *dir_results = opendir(path_results_c);
	if(ENOENT == errno){
		mkdir(path_results_c, ACCESSPERMS);
	}

	closedir(dir_results);

	path << folder << name << "/";
	const char* path_c = strdup(path.str().c_str());

	DIR *dir = opendir(path_c);
	if(ENOENT == errno){
		mkdir(path_c, ACCESSPERMS);
	}

	closedir(dir);

	filename << path.str() << name << std::setfill('0') << std::setw(ndigits) << n << ext;
	const char* filename_c = strdup(filename.str().c_str());

	checkCudaErrors(hipMemcpy(scalar_host, scalar_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

	FILE* fout = fopen(filename_c, "wb+");

	fwrite(scalar_host, 1, mem_size_scalar, fout);

	if(ferror(fout)){
		fprintf(stderr, "Error saving to %s\n", filename_c);
		perror("");
	}
	else{
		if(!quiet){
			printf("Saved to %s\n", filename_c);
		}
	}
	fclose(fout);
}

void wrapper_input(unsigned int *nx, unsigned int *ny, double *rho, double *u, double *nu, const double *tau){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Nx_d), nx, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Ny_d), ny, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(rho0_d), rho, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(u_max_d), u, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nu_d), nu, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), tau, sizeof(double)));
}

void wrapper_lattice(unsigned int *ndir, double *c, double *w_0, double *w_s, double *w_d){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(q), ndir, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cs_d), c, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(w0_d), w_0, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ws_d), w_s, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wd_d), w_d, sizeof(double)));
}

__host__ int* generate_e(int *e, std::string mode){

	int *temp_e;

	size_t mem_e = ndir*sizeof(int);

	checkCudaErrors(hipMalloc(&temp_e, mem_e));
	checkCudaErrors(hipMemcpy(temp_e, e, mem_e, hipMemcpyHostToDevice));

	if(mode == "x"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ex_d), &temp_e, sizeof(temp_e)));
	}
	else if(mode == "y"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ey_d), &temp_e, sizeof(temp_e)));
	}

	return temp_e;
}

__host__ bool* generate_mesh(bool *mesh, std::string mode){

	int mode_num;
	bool *temp_mesh;

	checkCudaErrors(hipMalloc(&temp_mesh, mem_mesh));
	checkCudaErrors(hipMemcpy(temp_mesh, mesh, mem_mesh, hipMemcpyHostToDevice));
	

	if(mode == "solid"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cylinder_d), &temp_mesh, sizeof(temp_mesh)));
		mode_num = 1;
	}
	else if(mode == "fluid"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fluid_d), &temp_mesh, sizeof(temp_mesh)));
		mode_num = 1;
	}

	if(meshprint){
		gpu_print_mesh<<< 1, 1 >>>(mode_num);
		printf("\n");
	}

	return temp_mesh;
}

__global__ void gpu_print_mesh(int mode){
	if(mode == 1){
		for(int y = 0; y < Ny_d; ++y){
			for(int x = 0; x < Nx_d; ++x){
				printf("%d ", cylinder_d[Nx_d*y + x]);
			}
		printf("\n");
		}
	}
	else if(mode == 2){
		for(int y = 0; y < Ny_d; ++y){
			for(int x = 0; x < Nx_d; ++x){
				printf("%d ", fluid_d[Nx_d*y + x]);
			}
		printf("\n");
		}
	}
}

__host__ void initialization(double *array, double value){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_initialization<<< grid, block >>>(array, value);
	getLastCudaError("gpu_print_array kernel error");
}

__global__ void gpu_initialization(double *array, double value){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	array[gpu_scalar_index(x, y)] = value;
}

__host__ bool* create_pinned_mesh(bool *array){

	bool *pinned;
	const unsigned int bytes = Nx*Ny*sizeof(bool);

	checkCudaErrors(hipHostMalloc((void**)&pinned, bytes));
	memcpy(pinned, array, bytes);
	return pinned;
}

__host__ double* create_pinned_double(){

	double *pinned;

	checkCudaErrors(hipHostMalloc((void**)&pinned, mem_size_scalar));
	return pinned;
}
