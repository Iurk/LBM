#include <stdio.h>
#include <stdlib.h>

#include "dados.h"
#include "saving.h"
#include "seconds.h"

#include "LBM.h"
#include "boundary.h"

using namespace myGlobals;

int main(int argc, char const *argv[]){

	// Writing Simulation Parameters
	printf("Simulating the Poiseulle Flow\n");
	printf("  Domain size: %ux%u\n", Nx, Ny);
	printf("           nu: %g\n", nu);
	printf("          tau: %g\n", tau);
	printf("        u_max: %g\n", u_max);
	printf("         rho0: %g\n", rho0);
	printf("           Re: %g\n", Re);
	printf("  Times Stpes: %u\n", NSTEPS);
	printf("   Save every: %u\n", NSAVE);
	printf("Message every: %u\n", NMSG);

	double bytesPerMiB = 1024.0*1024.0;
	double bytesPerGiB = 1024.0*1024.0*1024.0;

	// Getting Device Info and Writing then
	checkCudaErrors(hipSetDevice(0));
	int deviceId = 0;
	checkCudaErrors(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));

	size_t gpu_free_mem, gpu_total_mem;
	checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));

	printf("CUDA information\n");
	printf("      Using device: %d\n", deviceId);
	printf("              Name: %s\n", deviceProp.name);
	printf("   Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("     Global Memory: %.1f MiB\n", deviceProp.totalGlobalMem/bytesPerMiB);
	printf("       Free Memory: %.1f MiB\n", gpu_free_mem/bytesPerMiB);
	printf("\n");

	// Declaration and Allocation in device Memory
	double *f1_gpu, *f2_gpu, *feq_gpu, *fneq_gpu;
	double *rho_gpu, *ux_gpu, *uy_gpu, *ux_old_gpu;
	double *prop_gpu;

	checkCudaErrors(hipMalloc((void**)&f1_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&f2_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&feq_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&fneq_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&rho_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&uy_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_old_gpu, mem_size_scalar));

	const size_t mem_size_conv = 2*1*Ny/nThreads*sizeof(double);
	const size_t mem_size_props = Nx/nThreads*Ny*sizeof(double);
	checkCudaErrors(hipMalloc((void**)&prop_gpu, mem_size_props));
	checkCudaErrors(hipMalloc((void**)&conv_gpu, mem_size_conv));

	double *scalar_host, *conv_host;
	scalar_host = create_pinned_double();
	conv_host = create_pinned_double();
	if(scalar_host == NULL){
		fprintf(stderr, "Error: unable to allocate required memory (%.1f MiB).\n", mem_size_scalar/bytesPerMiB);
		exit(-1);
	}

	size_t total_mem_bytes = 3*mem_size_ndir + 3*mem_size_scalar + mem_size_props;
	
	// Creating Events for time measure
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	// Allocation of Input data in Device constant memory
	wrapper_input(&Nx, &Ny, &rho0, &u_max, &nu, &tau);

	// Allocation of Lattice data in Device constant and global memory
	wrapper_lattice(&ndir, &cs, &w0, &ws, &wd);

	int *ex_gpu, *ey_gpu;

	ex_gpu = generate_e(ex, "x");
	ey_gpu = generate_e(ey, "y");

	bool *solid_p, *fluid_p;
	bool *solid_gpu, *fluid_gpu;

	solid_p = create_pinned_mesh(cylinder);
	fluid_p = create_pinned_mesh(fluid);

	// Generating Mesh
	solid_gpu = generate_mesh(solid_p, "solid");
	fluid_gpu = generate_mesh(fluid_p, "fluid");

	// Initialization
	initialization(rho_gpu, rho0);
	initialization(ux_gpu, u_max);
	initialization(uy_gpu, 0.0);

	init_equilibrium(f1_gpu, rho_gpu, ux_gpu, uy_gpu);
	checkCudaErrors(hipMemset(fneq_gpu, 0, mem_size_ndir));

	save_scalar("rho",rho_gpu, scalar_host, 0);
	save_scalar("ux", ux_gpu, scalar_host, 0);
	save_scalar("uy", uy_gpu, scalar_host, 0);
	
	// Simulation Start
	double begin = seconds();
	checkCudaErrors(hipEventRecord(start, 0));

	double conv_error;
	unsigned int end_step;
	std::vector<double> fluid_prop;

	// Main Loop
	printf("Starting main loop...\n");
	std::cout << std::setw(10) << "Timestep" << std::setw(20) << "Convergence" << std::endl;
	for(unsigned int n = 0; n < NSTEPS; ++n){
		bool save = (n+1)%NSAVE == 0;
		bool msg = (n+1)%NMSG == 0;
		bool need_scalars = save || (msg);
/*
		double *ux_test;

		ux_test = (double*)malloc(mem_size_scalar);
		checkCudaErrors(hipMemcpy(ux_test, ux_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

		for(int y = 0; y < Ny; ++y){
			std::cout << y << "-> ";
			for(int x = 0; x < Nx; ++x){
				std::cout << ux_test[Nx*y+x] << " ";
			}
			std::cout << std::endl;
		}
*/
		stream_collide_save(f1_gpu, f2_gpu, feq_gpu, fneq_gpu, rho_gpu, ux_gpu, uy_gpu, need_scalars);

		if(save){
			save_scalar("rho",rho_gpu, scalar_host, n+1);
			save_scalar("ux", ux_gpu, scalar_host, n+1);
			save_scalar("uy", uy_gpu, scalar_host, n+1);
		}

		double *temp = f1_gpu;
		f1_gpu = f2_gpu;
		f2_gpu = temp;

		conv_error = report_convergence(n+1, ux_gpu, ux_old_gpu, conv_host, conv_gpu, msg);

		end_step = n+1;
		if(conv_error < erro_max && n > 2){
			break;
		}

		checkCudaErrors(hipMemcpy(ux_old_gpu, ux_gpu, mem_size_scalar, hipMemcpyDeviceToDevice));
	}

	bool msg = 0 == 0;
	std::cout << std::setw(10) << "Timestep" << std::setw(10) << "E" << std::setw(15) << "L2" << std::setw(23) << "Convergence" << std::endl;
	fluid_prop = report_flow_properties(end_step, conv_error, rho_gpu, ux_gpu, uy_gpu, prop_gpu, scalar_host, msg);
	save_terminal(end_step, conv_error, fluid_prop);
	
	// Measuring time
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float miliseconds = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&miliseconds, start, stop));

	double end = seconds();
	double runtime = end - begin;
	double gpu_runtime = 0.001*miliseconds;

	size_t doubles_read = ndir;
	size_t doubles_wirtten = ndir;
	size_t doubles_saved = 3;

	size_t nodes_updated = NSTEPS*size_t(Nx*Ny);
	size_t nodes_saved = (NSTEPS/NSAVE)*size_t(Nx*Ny);
	double speed = nodes_updated/(1e6*runtime);

	double bandwidth = (nodes_updated*(doubles_read + doubles_wirtten) + nodes_saved*(doubles_saved))*sizeof(double)/(runtime*bytesPerGiB);

	// Writing the performance
	printf("Performance Information\n");
	printf(" Memory Allocated (GPU): %.1f (MiB)\n", total_mem_bytes/bytesPerMiB);
	printf("Memory Allocated (host): %.1f (MiB)\n", mem_size_scalar/bytesPerMiB);
	printf("              Timesteps: %u\n", NSTEPS);
	printf("             Clock Time: %.3f (s)\n", runtime);
	printf("            GPU runtime: %.3f (s)\n", gpu_runtime);
	printf("                  Speed: %.2f (Mlups)\n", speed);
	printf("               Bandwith: %.1f (GiB/s)\n", bandwidth);

	// Cleaning up

	// Destroying Events
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	// Freeing Device and CPU Memory
	// LBM variables
	checkCudaErrors(hipFree(f1_gpu));
	checkCudaErrors(hipFree(f2_gpu));
	checkCudaErrors(hipFree(feq_gpu));
	checkCudaErrors(hipFree(fneq_gpu));
	checkCudaErrors(hipFree(rho_gpu));
	checkCudaErrors(hipFree(ux_gpu));
	checkCudaErrors(hipFree(uy_gpu));
	checkCudaErrors(hipFree(ux_old_gpu));
	checkCudaErrors(hipFree(prop_gpu));
	checkCudaErrors(hipFree(ex_gpu));
	checkCudaErrors(hipFree(ey_gpu));

	// Mesh arrays
	checkCudaErrors(hipFree(solid_gpu));
	checkCudaErrors(hipFree(fluid_gpu));
	checkCudaErrors(hipHostFree(solid_p));
	checkCudaErrors(hipHostFree(fluid_p));

	// Host arrays
	checkCudaErrors(hipHostFree(scalar_host));
	checkCudaErrors(hipHostFree(conv_host));

	hipDeviceReset();

	return 0;
}
