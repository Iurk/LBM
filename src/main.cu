#include <stdio.h>
#include <stdlib.h>

#include "seconds.h"
#include "LBM.h"
#include "dados.h"

using namespace myGlobals;

int main(int argc, char const *argv[]){

	// Writing Simulation Parameters
	printf("Simulating the Poiseulle Flow\n");
	printf("  Domain size: %ux%u\n", Nx, Ny);
	printf("           nu: %g\n", nu);
	printf("          tau: %g\n", tau);
	printf("        u_max: %g\n", u_max);
	printf("         rho0: %g\n", rho0);
	printf("           Re: %g\n", Re);
	printf("  Times Stpes: %u\n", NSTEPS);
	printf("   Save every: %u\n", NSAVE);
	printf("Message every: %u\n", NMSG);

	double bytesPerMiB = 1024.0*1024.0;
	double bytesPerGiB = 1024.0*1024.0*1024.0;

	// Getting Device Info and Writing then
	checkCudaErrors(hipSetDevice(0));
	int deviceId = 0;
	checkCudaErrors(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));

	size_t gpu_free_mem, gpu_total_mem;
	checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));

	printf("CUDA information\n");
	printf("      Using device: %d\n", deviceId);
	printf("              Name: %s\n", deviceProp.name);
	printf("   Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("     Global Memory: %.1f MiB\n", deviceProp.totalGlobalMem/bytesPerMiB);
	printf("       Free Memory: %.1f MiB\n", gpu_free_mem/bytesPerMiB);
	printf("\n");

	// Declaration and Allocation in device Memory
	double *f0_gpu, *f1_gpu, *f2_gpu;
	double *f0neq_gpu, *f1neq_gpu;
	double *rho_gpu, *ux_gpu, *uy_gpu;
	double *prop_gpu;

	checkCudaErrors(hipMalloc((void**)&f0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&f1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&f2_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&f0neq_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&f1neq_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&rho_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&uy_gpu, mem_size_scalar));

	const size_t mem_size_props = Nx/nThreads*Ny*sizeof(double);
	checkCudaErrors(hipMalloc((void**)&prop_gpu, mem_size_props));

	double *scalar_host;
	scalar_host = create_pinned_double();
	if(scalar_host == NULL){
		fprintf(stderr, "Error: unable to allocate required memory (%.1f MiB).\n", mem_size_scalar/bytesPerMiB);
		exit(-1);
	}

	size_t total_mem_bytes = mem_size_0dir + 2*mem_size_n0dir + 3*mem_size_scalar + mem_size_props;
	
	// Creating Events for time measure
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	// Declaration and Allocation of Input data in Device constant memory
	unsigned int *ptrNx, *ptrNy;
	double *ptrrho0, *ptru_max;
	double *ptrNu;
	const double *ptrTau;

	ptrNx = &Nx; ptrNy = &Ny;
	ptrrho0 = &rho0; ptru_max = &u_max;
	ptrNu = &nu; ptrTau = &tau;

	wrapper_input(ptrNx, ptrNy, ptrrho0, ptru_max, ptrNu, ptrTau);

	// Declaration and Allocation of Lattice data in Device constant and global memory
	unsigned int *ptrNdir;
	double *ptrcs, *ptrW0, *ptrWs, *ptrWd;

	ptrNdir = &ndir; ptrcs = &cs; 
	ptrW0 = &w0; ptrWs = &ws; ptrWd = &wd;

	wrapper_lattice(ptrNdir, ptrcs, ptrW0, ptrWs, ptrWd);

	int *ex_gpu, *ey_gpu;

	ex_gpu = generate_e(ex, "x");
	ey_gpu = generate_e(ey, "y");

	bool *solid_p, *fluid_p;
	bool *solid_gpu, *fluid_gpu;

	solid_p = create_pinned_mesh(cylinder);
	fluid_p = create_pinned_mesh(fluid);

	// Generating Mesh
	solid_gpu = generate_mesh(solid_p, "solid");
	fluid_gpu = generate_mesh(fluid_p, "fluid");

	// Initialization
	initialization(rho_gpu, rho0);
	initialization(ux_gpu, u_max);
	initialization(uy_gpu, 0.0);

	init_equilibrium(f0_gpu, f1_gpu, rho_gpu, ux_gpu, uy_gpu);
	checkCudaErrors(hipMemset(f0neq_gpu, 0, mem_size_0dir));
	checkCudaErrors(hipMemset(f1neq_gpu, 0, mem_size_n0dir));

	save_scalar("rho",rho_gpu, scalar_host, 0);
	save_scalar("ux", ux_gpu, scalar_host, 0);
	save_scalar("uy", uy_gpu, scalar_host, 0);
	
	// Simulation Start
	double begin = seconds();
	checkCudaErrors(hipEventRecord(start, 0));

	// Main Loop
	for(unsigned int n = 0; n < NSTEPS; ++n){
		bool save = (n+1)%NSAVE == 0;
		bool msg = (n+1)%NMSG == 0;
		bool need_scalars = save || (msg && computeFlowProperties);
/*
		double *ux_test;

		ux_test = (double*)malloc(mem_size_scalar);
		checkCudaErrors(hipMemcpy(ux_test, ux_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

		for(int y = 0; y < Ny; ++y){
			std::cout << y << "-> ";
			for(int x = 0; x < Nx; ++x){
				std::cout << ux_test[Nx*y+x] << " ";
			}
			std::cout << std::endl;
		}
*/
		stream_collide_save(f0_gpu, f1_gpu, f2_gpu, f0neq_gpu, f1neq_gpu, rho_gpu, ux_gpu, uy_gpu, need_scalars);

		if(save){
			save_scalar("rho",rho_gpu, scalar_host, n+1);
			save_scalar("ux", ux_gpu, scalar_host, n+1);
			save_scalar("uy", uy_gpu, scalar_host, n+1);
		}

		double *temp = f1_gpu;
		f1_gpu = f2_gpu;
		f2_gpu = temp;

		if(msg){
			if(computeFlowProperties){
				report_flow_properties(n+1, rho_gpu, ux_gpu, uy_gpu, prop_gpu, scalar_host);
			}

			if(!quiet){
				printf("Completed timestep %d\n", n+1);
			}
		}
	}
	
	// Measuring time
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float miliseconds = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&miliseconds, start, stop));

	double end = seconds();
	double runtime = end - begin;
	double gpu_runtime = 0.001*miliseconds;

	size_t doubles_read = ndir;
	size_t doubles_wirtten = ndir;
	size_t doubles_saved = 3;

	size_t nodes_updated = NSTEPS*size_t(Nx*Ny);
	size_t nodes_saved = (NSTEPS/NSAVE)*size_t(Nx*Ny);
	double speed = nodes_updated/(1e6*runtime);

	double bandwidth = (nodes_updated*(doubles_read + doubles_wirtten) + nodes_saved*(doubles_saved))*sizeof(double)/(runtime*bytesPerGiB);

	// Writing the performance
	printf("Performance Information\n");
	printf(" Memory Allocated (GPU): %.1f (MiB)\n", total_mem_bytes/bytesPerMiB);
	printf("Memory Allocated (host): %.1f (MiB)\n", mem_size_scalar/bytesPerMiB);
	printf("              Timesteps: %u\n", NSTEPS);
	printf("             Clock Time: %.3f (s)\n", runtime);
	printf("            GPU runtime: %.3f (s)\n", gpu_runtime);
	printf("                  Speed: %.2f (Mlups)\n", speed);
	printf("               Bandwith: %.1f (GiB/s)\n", bandwidth);

	// Cleaning up

	// Destroying Events
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	// Freeing Device and CPU Memory
	// LBM variables
	checkCudaErrors(hipFree(f0_gpu));
	checkCudaErrors(hipFree(f1_gpu));
	checkCudaErrors(hipFree(f2_gpu));
	checkCudaErrors(hipFree(f0neq_gpu));
	checkCudaErrors(hipFree(f1neq_gpu));
	checkCudaErrors(hipFree(rho_gpu));
	checkCudaErrors(hipFree(ux_gpu));
	checkCudaErrors(hipFree(uy_gpu));
	checkCudaErrors(hipFree(prop_gpu));
	checkCudaErrors(hipFree(ex_gpu));
	checkCudaErrors(hipFree(ey_gpu));

	// Mesh arrays
	checkCudaErrors(hipFree(solid_gpu));
	checkCudaErrors(hipFree(fluid_gpu));
	checkCudaErrors(hipHostFree(solid_p));
	checkCudaErrors(hipHostFree(fluid_p));

	// Host arrays
	checkCudaErrors(hipHostFree(scalar_host));

	hipDeviceReset();

	return 0;
}
